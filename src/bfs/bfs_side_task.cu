#include "hip/hip_runtime.h"
#include "bfs.h"
#include <argparse/argparse.hpp>
#include <iostream>
#include "task.h"
#include <thread>
#include <unistd.h>
#include <csignal>
#include <grpcpp/security/server_credentials.h>
#include <grpcpp/server.h>
#include <grpcpp/server_builder.h>
#include <atomic>
#include "cutil_subset.h"
#include "worklistc.h"
#include <chrono>

__global__ void insert(int source, Worklist2 queue) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id == 0) queue.push(source);
  return;
}

__global__ void bfs_kernel(int m, const uint64_t *row_offsets, 
                           const IndexT *column_indices, 
                           DistT *dists, Worklist2 in_queue, 
                           Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if (in_queue.pop_id(tid, src)) {
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src+1];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			if ((dists[dst] == MYINFINITY) && 
          (atomicCAS(&dists[dst], MYINFINITY, dists[src]+1) == MYINFINITY)) {
				assert(out_queue.push(dst));
			}
		}
	}
}

class BfsLinearSideTask final : public BubbleBanditTask {
 private:
  bool with_profiler_;
  std::string file_type_;
  std::string graph_prefix_;
  std::string symmetrize_;
  std::string reverse_;
  std::string source_id_;
  std::atomic<bool> init_event_;
  std::atomic<bool> start_event_;
  std::atomic<bool> pause_event_;
  std::atomic<bool> stop_event_;
  std::atomic<bool> preempt_event_;
  std::atomic<int> counter_;
  std::atomic<int64_t> ts0_;
  std::atomic<int64_t> ts1_;
  std::thread runner_;
  double duration_;
  std::atomic<double> end_time_;

  Graph *g_ptr;
  VertexId m;
  size_t nnz;
  uint64_t *h_row_offsets;
  VertexId *h_column_indices;
  uint64_t *d_row_offsets;
  VertexId *d_column_indices;
  std::vector<DistT> h_dists;

  int iter;
  int item_num;
  int thread_num;
  int block_num;
  Worklist2 queue1;
  Worklist2 queue2;
  Worklist2 *in_frontier, *out_frontier;

  DistT zero = 0;
  DistT * d_dists;



    auto do_i_have_enough_time() -> bool {
    // Get the current time in microseconds
    auto current_time = get_current_time_in_micro();

    // Check if the current time is less than the end time
    return end_time_ - current_time > duration_;
  
  }

 public:
  BfsLinearSideTask(int task_id, std::string name, std::string device, std::string scheduler_addr, bool with_profiler,
  std::string file_type, std::string graph_prefix, std::string symmetrize, std::string reverse, std::string source_id) 
  : BubbleBanditTask(task_id, name, device, scheduler_addr) {
    with_profiler_ = with_profiler;
    file_type_ = file_type;
    graph_prefix_ = graph_prefix;
    symmetrize_ = symmetrize;
    reverse_ = reverse;
    source_id_ = source_id;
    init_event_ = false;
    start_event_ = false;
    pause_event_ = false;
    stop_event_ = false;
    preempt_event_ = false;
    counter_ = 0;
    ts0_ = 0;
    ts1_ = 0;
    duration_ = 0.1;
    end_time_ = 0.0;
  }

  int64_t init(int64_t task_id) override {
    assert(task_id == task_id_);
    std::cout << "Init task " << task_id << std::endl;
    init_event_ = true;
    return 0;
  }

  int64_t start(int64_t task_id, double end_time) override {
    assert(task_id == task_id_);
    std::cout << "Start task " << task_id << " with end time " << end_time << std::endl;
    end_time_ = end_time;
    start_event_ = true;
    return 0;
  }

  int64_t pause(int64_t task_id) override {
    assert(task_id == task_id_);
    std::cout << "Pause task " << task_id << std::endl;
    pause_event_ = true;
    return 0;
  }

  int64_t stop(int64_t task_id) override {
    assert(task_id == task_id_);
    std::cout << "Stop task " << task_id << std::endl;
    stop_event_ = true;
    runner_.join();
    std::cout << "Task " << task_id << " stopped" << std::endl;
    kill(getpid(), SIGINT);
    return 0;
  }

  int64_t preempt(int64_t task_id) override {
    assert(task_id == task_id_);
    std::cout << "Preempt task " << task_id << std::endl;
    preempt_event_ = true;
    return 0;
  }
  

  auto submitted_to_created() -> void override {
    auto device = device_.at(5) - '0';
    std::cout << "Device: " << device << std::endl;
    hipSetDevice(device);
    g_ptr = new Graph(graph_prefix_, file_type_, std::stoi(symmetrize_), 1);
    auto &g = *g_ptr;
    
    int source = std::stoi(source_id_);
    m = g.V();
    std::vector<DistT> distances(m, MYINFINITY);
    h_dists = &distances[0];
    
    state_ = BubbleBanditTask::State::CREATED;
    
    nnz = g.E();
    h_row_offsets = g.out_rowptr();
    h_column_indices = g.out_colidx();
    
    *d_row_offsets;
    *d_column_indices;
    zero = 0;
    std::cout << "Max size: " << m << std::endl;
    queue1 = Worklist2(m);
    queue2 = WorkList2(m);
    *in_frontier = &queue1, *out_frontier = &queue2;
    iter = 0;
    item_num = 1;
    thread_num = BLOCK_SIZE;
    block_num = (m - 1) / thread_num + 1;
    printf("Launching CUDA BFS solver (%d threads/CTA) ...\n", thread_num);
  }
  
  auto created_to_paused() -> void override {
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    
    CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(uint64_t)));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(VertexId)));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;

    CUDA_SAFE_CALL(hipMalloc((void **)&d_dists, m * sizeof(DistT)));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipMemcpy(d_dists, h_dists, m * sizeof(DistT), hipMemcpyHostToDevice));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipMemcpy(&d_dists[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    insert<<<1, thread_num>>>(source, *in_frontier);
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    item_num = in_frontier->nitems();
    std::cout << __FILE__ << ":" << __LINE__ << std::endl;
    state = BubbleBanditTask::State::PENDING;
  }
          break;
        }
        case BubbleBanditTask::State::PENDING: {
          if (start_event_) {
            start_event_ = false;
            state = BubbleBanditTask::State::RUNNING;
            std::cout << "State from PENDING to RUNNING" << std::endl;
          } else if (preempt_event_) {
            preempt_event_ = false;
            // TODO: Fully clear GPU memory.
            CUDA_SAFE_CALL(hipMemcpy(h_dists, d_dists, m * sizeof(DistT), hipMemcpyDeviceToHost))
            CUDA_SAFE_CALL(hipFree(d_row_offsets))
            CUDA_SAFE_CALL(hipFree(d_column_indices))
            CUDA_SAFE_CALL(hipFree(d_dists))

            state = BubbleBanditTask::State::CREATED;
            std::cout << "State from PENDING to CREATED" << std::endl;
          }
          break;
        }
        case BubbleBanditTask::State::RUNNING: {
          if (pause_event_) {
            pause_event_ = false;
            state = BubbleBanditTask::State::PENDING;
            std::cout << "State from RUNNING to PENDING" << std::endl;
          } else if (preempt_event_) {
            preempt_event_ = false;
            CUDA_SAFE_CALL(hipMemcpy(h_dists, d_dists, m * sizeof(DistT), hipMemcpyDeviceToHost))
            CUDA_SAFE_CALL(hipFree(d_row_offsets))
            CUDA_SAFE_CALL(hipFree(d_column_indices))
            CUDA_SAFE_CALL(hipFree(d_dists))
            state = BubbleBanditTask::State::CREATED;
            std::cout << "State from RUNNING to CREATED" << std::endl;
          } else {
            if (!do_i_have_enough_time()) {
              printf("I do not have enough time, current: %f, end: %f\n", get_current_time_in_micro(), end_time_.load());
              auto end_time = end_time_.load();
              if (end_time - get_current_time_in_micro() > 1000) {
                usleep((end_time - get_current_time_in_micro()) / 1000);
              }
            } else {
              ++ iter;
              block_num = (item_num - 1) / thread_num + 1;
              std::cout << "iteration " << iter << ": frontier_size = " << item_num << std::endl;
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
              bfs_kernel<<<block_num, thread_num>>> (m, d_row_offsets, d_column_indices,
                                                      d_dists, *in_frontier, *out_frontier);
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
//              CUDA_SAFE_CALL(hipDeviceSynchronize())
              item_num = out_frontier->nitems();
              std::cout << "New frontier_size = " << item_num << std::endl;
//              CUDA_SAFE_CALL(hipDeviceSynchronize())
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
              Worklist2 *tmp = in_frontier;
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
              in_frontier = out_frontier;
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
              out_frontier = tmp;
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
              out_frontier->reset();
              std::cout << __FILE__ << ": "<< __LINE__ << std::endl;
//              CUDA_SAFE_CALL(hipDeviceSynchronize())
              if (item_num <= 0) {
                // TODO: clean up.
                goto BREAK_LOOP;
              }
            }
          }
          break;
        }
        default: {
          assert(false);
        }
      }
      if (stop_event_) {
        break;
      }
      usleep(10000);
    }
BREAK_LOOP:

    if (with_profiler_) {
      // TODO: You probably want to add some profiling logic here.
    }
    if (stop_event_) {
      stop_event_ = false;
    } else {
      scheduler_client_.finish_task(task_id_);
    }
  }

  void finish() override {
  }

  void start_runner() override {
    std::cout << "Start runner of task " << task_id_ << std::endl;
    runner_ = std::thread([this] { run(); });
  }
};

grpc::Server *server_ptr;

void signalHandler(int signum) {
  std::cout << "Interrupt signal (" << signum << ") received.\n";

  // cleanup and close up stuff here
  // terminate program
  server_ptr->Shutdown();

  exit(signum);  
}

int main(int argc, char **argv) {
  argparse::ArgumentParser program("program_name");
  program.add_argument("-n", "--name");
  program.add_argument("-s", "--scheduler_addr");
  program.add_argument("-i", "--task_id");
  program.add_argument("-d", "--device");
  program.add_argument("-a", "--addr");
  // TODO: Jiashu: Fix profiler flag
  // program.add_argument("-p", "--profiler");
  program.add_argument("-t", "--file_type");
  program.add_argument("-g", "--graph_prefix");
  program.add_argument("--symmetrize");
  program.add_argument("--reverse");
  program.add_argument("--source_id");

  try {
    program.parse_args(argc, argv);
  }
  catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  auto name = program.get<std::string>("--name");
  auto scheduler_addr = program.get<std::string>("--scheduler_addr");
  auto task_id = std::stoi(program.get<std::string>("--task_id"));
  auto device = program.get<std::string>("--device");
  auto addr = program.get<std::string>("--addr");
  // auto with_profiler = bool(std::stoi(program.get<std::string>("--profiler")));
  auto with_profiler = false;
  auto file_type = program.get<std::string>("--file_type");
  auto graph_prefix = program.get<std::string>("--graph_prefix");
  auto symmetrize = program.get<std::string>("--symmetrize");
  auto reverse = program.get<std::string>("--reverse");
  auto source_id = program.get<std::string>("--source_id");

  auto task = BfsLinearSideTask(task_id, name, device, scheduler_addr, with_profiler, 
  file_type, graph_prefix, symmetrize, reverse, source_id);

  // task.init(task_id);
  // task.run();
  // task.stop(task_id);
  auto service = TaskServiceImpl(&task);

  grpc::ServerBuilder builder;
  builder.AddListeningPort(addr, grpc::InsecureServerCredentials());
  builder.RegisterService(&service);
  std::unique_ptr<grpc::Server> server(builder.BuildAndStart());
  std::cout << "Server listening on " << addr << std::endl;
  task.start_runner();
  server->Wait();

  signal(SIGINT, signalHandler);

  return 0;
}
